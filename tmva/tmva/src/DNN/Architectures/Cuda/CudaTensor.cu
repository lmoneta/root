// @(#)root/tmva/tmva/dnn:$Id$
// Author: Simon Pfreundschuh 13/07/16

/*************************************************************************
 * Copyright (C) 2016, Simon Pfreundschuh                                *
 * All rights reserved.                                                  *
 *                                                                       *
 * For the licensing terms see $ROOTSYS/LICENSE.                         *
 * For the list of contributors see $ROOTSYS/README/CREDITS.             *
 *************************************************************************/

/////////////////////////////////////////////
// Implementation of the TCudaTensor class. //
/////////////////////////////////////////////

#include "TMVA/DNN/Architectures/Cuda/CudaTensor.h"
#include "TMVA/DNN/Architectures/Cuda/Device.h"

#include <algorithm>
#include <cassert>
#include <iostream>

namespace TMVA {
namespace DNN  {


// Static members.
//____________________________________________________________________________
/*template<typename AFloat>
size_t                   TCudaTensor<AFloat>::fInstances        = 0;*/
/*template<typename AFloat>
hipblasHandle_t           TCudaTensor<AFloat>::fCublasHandle     = nullptr;*/
/*template<typename AFloat>
hipdnnHandle_t            TCudaTensor<AFloat>::fCudnnHandle      = nullptr;*/
template<typename AFloat>
std::vector<hipdnnHandle_t> TCudaTensor<AFloat>::fCudnnHandle(1);
/*template<typename AFloat>
hipdnnTensorDescriptor_t  TCudaTensor<AFloat>::fTensorDescriptor = nullptr;*/
template<typename AFloat>
hipdnnDataType_t          TCudaTensor<AFloat>::fDataType         = HIPDNN_DATA_FLOAT;
/*template<typename AFloat>
AFloat                   * TCudaTensor<AFloat>::fDeviceReturn   = nullptr;*/
/*template<typename AFloat>
AFloat                   * TCudaTensor<AFloat>::fOnes           = nullptr;*/
/*template<typename AFloat>
hiprandState_t            * TCudaTensor<AFloat>::fCurandStates   = nullptr;*/
/*template<typename AFloat>
size_t                   TCudaTensor<AFloat>::fNCurandStates    = 0;*/
/*template<typename AFloat>
size_t                   TCudaTensor<AFloat>::fNOnes            = 0;*/
/*template<typename AFloat>
std::vector<std::vector<int> >         TCudaTensor<AFloat>::fStreamIndxs(std:vector<int>(), std::vector<int>());*/
template<typename AFloat>
std::vector<int>         TCudaTensor<AFloat>::fInstances(1,0);

/// This information is needed for the multi-dimensional indexing. See here:
/// https://en.wikipedia.org/wiki/Row-_and_column-major_order
/// https://docs.scipy.org/doc/numpy/reference/generated/numpy.ndarray.strides.html
template<typename AFloat>
std::vector<std::size_t> TCudaTensor<AFloat>::ComputeStridesFromShape(const std::vector<std::size_t> &shape, 
   bool rowmajorLayout)
{
   const auto size = shape.size();
   std::vector<std::size_t> strides(size);
   if (rowmajorLayout)  {
      for (std::size_t i = 0; i < size; i++) {
         if (i == 0) {
            strides[size - 1 - i] = 1;
         } else {
            strides[size - 1 - i] = strides[size - 1 - i + 1] * shape[size - 1 - i + 1];
         }
      }
   } else  {
      for (std::size_t i = 0; i < size; i++) {
         if (i == 0) {
            strides[i] = 1;
         } else {
            strides[i] = strides[i - 1] * shape[i - 1];
         }
      }
   }
   return strides;
}

// Constructors.
//____________________________________________________________________________
template<typename AFloat>
TCudaTensor<AFloat>::TCudaTensor()
    : fShape(), fStrides(), fNDim(0), fSize(0), fElementBuffer(), fStreamIndx(0), fTensorDescriptor(nullptr)
{
   InitializeCuda();
}


//____________________________________________________________________________
template<typename AFloat>
TCudaTensor<AFloat>::TCudaTensor(const std::vector<size_t> & shape,
                                 TCudaTensor::MemoryLayout layout,
                                 int device, int streamIndx)
    : fShape(shape), fStrides(shape.size()), fNDim(shape.size()), fDevice(device), fStreamIndx(streamIndx),
      fTensorDescriptor(nullptr), fMemoryLayout(layout)
{
   fStrides = ComputeStridesFromShape(fShape, layout==MemoryLayout::RowMajor);
   
   fSize = (layout==MemoryLayout::RowMajor) ? fStrides.front()*fShape.front() : 
                                              fStrides.back()*fShape.back(); 

   fElementBuffer = TCudaDeviceBuffer<AFloat>(fSize, 0);
   
   InitializeCuda();
}

//____________________________________________________________________________
template<typename AFloat>
TCudaTensor<AFloat>::TCudaTensor(const AFloat * host_data, const std::vector<size_t> & shape,
                                 TCudaTensor::MemoryLayout layout,
                                 int device, int streamIndx)
   : TCudaTensor(shape, layout, device, streamIndx)
{
   // do I need to allocate this buffer ???? 
   // is not a mem leak
   // AFloat * buffer = new AFloat[fSize];
   // size_t index = 0;
   // for (size_t j = 0; j < fSize; ++j) {
   //       buffer[j] = static_cast<AFloat>(host_data[j]);
   //    }
   // }

   hipMemcpy(fElementBuffer, host_data, fSize * sizeof(AFloat),
              hipMemcpyHostToDevice);
}

//____________________________________________________________________________
template<typename AFloat>
TCudaTensor<AFloat>::TCudaTensor(TCudaDeviceBuffer<AFloat> buffer,  
                                 const std::vector<size_t> & shape,
                                 TMVA::Experimental::MemoryLayout layout,
                                 int device, int streamIndx)
   : fNDim(shape.size()), fElementBuffer(buffer), fShape(shape), fStrides( shape.size()), fDevice(device), 
     fStreamIndx(streamIndx), fTensorDescriptor(nullptr), fMemoryLayout(layout)
{
   fStrides = ComputeStridesFromShape(fShape, layout==MemoryLayout::RowMajor);
   
   fSize = (layout==MemoryLayout::RowMajor) ? fStrides.front()*fShape.front() : 
                                              fStrides.back()*fShape.back();  
   InitializeCuda();  
}

//____________________________________________________________________________
//FIXME: Go to shared_ptr implementation of instance tracking
template <typename AFloat>
TCudaTensor<AFloat>::TCudaTensor(const TCudaTensor<AFloat>& oldTensor, size_t /*dim*/) :
   TCudaTensor(oldTensor.fShape, oldTensor.fMemoryLayout, oldTensor.fDevice, oldTensor.fStreamIndx)
{
   // No deep copy
   fStrides       = oldTensor.fStrides;
   fElementBuffer = oldTensor.fElementBuffer;   
        
   InitializeCuda();
}

//____________________________________________________________________________
template <typename AFloat>
TCudaTensor<AFloat>::TCudaTensor(const TCudaMatrix<AFloat>& matrix, size_t dim) :
   TCudaTensor( matrix.GetDeviceBuffer(), {matrix.GetNrows(), matrix.GetNcols()}, MemoryLayout::ColumnMajor)
{
   // No deep copy
   if (dim > 2) {
      // change shape from (nrows,ncols) to (nrows,ncols,1,1)
      // this works onlt for coolum major layout since this is same of TCudaMatrix
      fShape.insert(fShape.end(), dim-2, 1);
      fStrides.insert(fStrides.end(),dim-2,fSize); 
      fNDim = dim; 
   }

   InitializeCuda();
}

//____________________________________________________________________________
template <typename AFloat>
TCudaTensor<AFloat>::~TCudaTensor() 
{
   if (fTensorDescriptor && fTensorDescriptor.use_count() == 1 ) { 
      CUDNNCHECK(hipdnnDestroyTensorDescriptor(fTensorDescriptor->fCudnnDesc));
   }
   fInstances[fStreamIndx]--;

   // When all tensors in a streamIndx are destroyed, release cudnn resources 
   //if (--fInstances[fStreamIndx] <= 0) CUDNNCHECK(hipdnnDestroy(fCudnnHandle[fStreamIndx]));
//#endif
}

template<typename AFloat>
TCudaTensor<AFloat>::operator TMatrixT<AFloat>() const
{
   // this should work only for size 2 or 4 tensors
   if (fNDim < 4) {
      TCudaMatrix<AFloat> temp = GetMatrix();
      return temp;
   }
   // we can convert directy to TMatrix 
   assert(fNDim == 4); 
   size_t nRows = fShape[0]*fShape[1];
   size_t nCols = fShape[2]*fShape[3]; 
   TMatrixT<AFloat> hostMatrix( nRows, nCols ); 

   
   hipMemcpy(hostMatrix.GetMatrixArray(), fElementBuffer, fSize * sizeof(AFloat),
           hipMemcpyDeviceToHost);

   return hostMatrix;
}
//____________________________________________________________________________
template <typename AFloat>
inline void TCudaTensor<AFloat>::InitializeCuda()
{
   // If fNDim >= 4, a cudnn tensor is required and we initialize cudnn
   if (fNDim >= 2 && fSize > 0) {
      // Also check whether a new streamIndx has been opened
      if (fInstances.size() - 1 < fStreamIndx) {
         // If need to resize once, need probably to resize more often
         fInstances.resize(2*fStreamIndx + 1, 0);
         fCudnnHandle.resize(2*fStreamIndx + 1, nullptr);
      }
      if (fInstances[fStreamIndx] == 0) {
        CUDNNCHECK(hipdnnCreate(&fCudnnHandle[fStreamIndx]));
        //CUDNNCHECK(hipdnnSetStream(fCudnnHandle[fStreamIndx], fElementBuffer.GetComputeStream()));
        
        //hipblasCreate(&fCublasHandle);
        //CUDACHECK(hipMalloc(& fDeviceReturn, sizeof(AFloat)));
        //CUDACHECK(hipMalloc(& fCurandStates, TDevice::NThreads(*this)));
      }
      // if (TDevice::NThreads(*this) > (int) fNCurandStates) {
      //     fNCurandStates = TDevice::NThreads(*this);
      //     if (fCurandStates) {
      //         hipFree(fCurandStates);
      //     }
      //     hipMalloc(&fCurandStates, TDevice::NThreads(*this) * sizeof(hiprandState_t));
      //     InitializeCurandStates();
      // }

      if (!fTensorDescriptor) { 
         fTensorDescriptor = std::make_shared<TensorDescriptor>();
         CUDNNCHECK(hipdnnCreateTensorDescriptor(&(fTensorDescriptor->fCudnnDesc)));
      }
        
      fInstances[fStreamIndx]++;
      
      // Prevent template specialization of entire class
      if      (std::is_same<AFloat, double>::value) {fDataType = HIPDNN_DATA_DOUBLE;}
      else if (std::is_same<AFloat, float>::value)  {fDataType = HIPDNN_DATA_FLOAT;}

      SetTensorDescriptor();
   }

}
template<typename AFloat>
void TCudaTensor<AFloat>::SetTensorDescriptor() {
      if (!fTensorDescriptor) return; 
      if (fSize == 0) return;

      // cuDNN NdTensor format has a minsize of 4 tensor dimensions
      // 4D tensor is more performant on lower dimensions and supports all folowing operations
      //if (fNDim == 4) {
      Shape_t shape = fShape; 
      if (fNDim < 4 && fNDim > 1 ) { 
         // add 1 to tensor 
         if (fMemoryLayout == MemoryLayout::RowMajor)  
            shape.insert(shape.end(),4-fNDim, 1);
         else 
            shape.insert(shape.begin(),4-fNDim,1);
      } else if (fNDim > 4) { 
         std::cout << "Error : Dim = "<< fNDim 
         <<". Currently only 4D tensors are supported for the TMVA cuDNN backend."
         << std::endl;
      }
      if (fMemoryLayout == MemoryLayout::RowMajor)  {
            CUDNNCHECK(hipdnnSetTensor4dDescriptor(fTensorDescriptor->fCudnnDesc,
                                               HIPDNN_TENSOR_NCHW,// Layout of the tensor in memory
                                               fDataType,
                                               (int)shape[0],   // batch size
                                               (int)shape[1],   // no. channels
                                               (int)shape[2],   // image height
                                               (int)shape[3])); // image width
      }
      else {
            CUDNNCHECK(hipdnnSetTensor4dDescriptor(fTensorDescriptor->fCudnnDesc,
                       HIPDNN_TENSOR_NCHW,// Layout of the tensor in memory
                       fDataType,
                       (int)shape[3],   // batch size
                       (int)shape[2],   // no. channels
                       (int)shape[1],   // image height
                       (int)shape[0])); // image width
      }
      
      // Some operations in cudnn may not work with this tensor description
      //else if 
 
        /*CUDNNCHECK(hipdnnSetTensorNdDescriptor(fTensorDescriptor,
                                              fDataType,
                                              (int)fNDim,
                                              (int *)fShape.data(),
                                              (int *)fStrides.data()));*/
      //}
   
#ifdef NDEBUG
      size_t tensorSize;
      CUDNNCHECK(cudnnGetTensorSizeInBytes(fTensorDescriptor->fCudnnDesc, &tensorSize));
      assert(fSize == tensorSize/sizeof(AFloat));

        //    int n,c,h,w = 0; 
   // int s1,s2,s3,s4 = 0; 
   // hipdnnDataType_t  dataType; 
   // hipdnnGetTensor4dDescriptor( fTensorDescriptor, &dataType,&n,&c,&h,&w,&s1,&s2,&s3,&s4 );
   // std::vector<size_t>  shape_input = {n,c,h,w}; 
   // assert (shape_input == GetShape());

#endif

 
   }

//____________________________________________________________________________
template<typename AFloat>
void TCudaTensor<AFloat>::InitializeCurandStates()
{
   // dim3 blockDims = TDevice::BlockDims2D();
   // dim3 gridDims  = TDevice::GridDims2D(*this);
   // CurandInitializationKernel<<<gridDims, blockDims>>>(time(nullptr), fCurandStates);
}

template<typename AFloat>
void TCudaTensor<AFloat>::Print(const char * name, bool truncate) const
{
      //TCudaBuffer<AFloat> hostBuffer (fSize);
      //fElementBuffer.CopyTo(hostBuffer);
    #if 0  
      AFloat hostBuffer[fSize]; 

      hipMemcpy(hostBuffer, fElementBuffer, fSize * sizeof(AFloat),
                 hipMemcpyDeviceToHost);
      
      for (size_t i = 0; i < fSize; i++) std::cout << hostBuffer[i] << "  ";
   #endif
   PrintShape(name);
   size_t n = fSize; 
   if (n > 10 && truncate) n = 10; 
   std::cout << "Data : { ";
   for (size_t i = 0; i < n; ++i ) {
      AFloat * elementPointer = fElementBuffer + i; 
      std::cout << AFloat( TCudaDeviceReference<AFloat>(elementPointer) );
      if (i < n-1) std::cout << " , "; 
   }
   if (n < fSize) std::cout << "............   } "; 
   std::cout << " } " << std::endl;
}
template<typename AFloat>
void TCudaTensor<AFloat>::PrintShape(const char * name) const
{
      std::string memlayout = (GetLayout() == MemoryLayout::RowMajor) ? "RowMajor" : "ColMajor"; 
      std::cout << name << " shape : { ";
      for (size_t i = 0; i < fNDim-1; ++i ) 
         std::cout << fShape[i] << " , ";
      std::cout << fShape.back() << " } " << " Layout : " << memlayout << std::endl;
}
#if 0
// Conversion to RTensor
//____________________________________________________________________________
template<typename AFloat>
TCudaTensor<AFloat>::operator Experimental::RTensor<AFloat>() const
{
   std::vector<size_t> shape(fNDims, fNDims + fDim)
   
   Experimental::RTensor<AFloat> hostTensor( shape)

   AFloat * buffer = new AFloat[fSize];
   hipMemcpy(buffer, fElementBuffer, fSize * sizeof(AFloat),
              hipMemcpyDeviceToHost);

   int index = 0;
   for (int j = 0; j < fSize; j++) {
         hostTensor.GetData()[j] = static_cast<AFloat>(buffer[j]);
      }
   }

   delete[] buffer;
   return hostTensor;
}
#endif
// Explicit Instantiations.

template class TCudaTensor<float>;
template class TCudaTensor<double>;

} // namespace DNN
} // namespace TMVA
